#include <iostream>
#include <chrono>
#include "launcher.h"
#include "measure.h"
using namespace std;
using namespace pmkl;
using namespace pmkl::utils;

#define FLOAT_N 4
template <typename T, int vec_size>
struct alignas(sizeof(T) * vec_size) aligned_array {
    T val[vec_size];
    DEVICE T &operator[](int i) {
        return val[i];
    }
    DEVICE T const &operator[](int i) const {
        return val[i];
    }
};
typedef aligned_array<float, FLOAT_N> floatn;

void print_info(bool enable, size_t N, uint64_t total_bytes, double timems) {
    if (enable)
    cout << "{'numel':" << N << ", 'MBytes':" << total_bytes/1024/1024 << ", 'timems':" << timems << ", 'GBps':" << (total_bytes/(timems/1000))/1024/1024/1024 << "}" << endl;
}

template<typename T>
void eu_memset(T *out, size_t N, bool verbose = true) {
    int group_size = 256;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    auto l = GpuLauncher::GetInstance();
    l->stream_begin();
    hipEventRecord(start);
    l->submit(
        0, {(int)((N + group_size - 1) / group_size)}, {group_size},
        [=] DEVICE(KernelInfo &info) {
            auto idx = info.thread_idx(0) + info.thread_range(0) * info.block_idx(0);
            if(idx < N) {
                floatn temp;
#pragma unroll
                for(int i=0; i<FLOAT_N; i++)
                    temp[i] = 1;
                out[idx] = temp;
            }
        });
    hipEventRecord(stop);
    l->stream_sync();
    hipEventSynchronize(stop);
    l->stream_end();
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    print_info(verbose, N, 1*N*sizeof(T), ms);
}

int main() {
    std::cout << "-------------------- output --------------------\n";
    auto l = GpuLauncher::GetInstance();
    auto d = sizeof(floatn);
    int numel = 256*1024*1024/d;
    auto out = l->malloc<floatn>(numel);
    
    for(int numel = 1*1024*1024/d; numel < 256*1024*1024/d; numel += 1*1024*1024/d) {
        eu_memset<floatn>(out, numel);
    }

    l->free(out);
}
